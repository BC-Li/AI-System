#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__global__ void matmul_kernel(
    const scalar_t* A,
    const scalar_t* B,
    scalar_t* C,
    const int M, 
    const int K, 
    const int N,
    const bool trans_A = false,
    const bool trans_B = false) 
{
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < N)
    {
        scalar_t sum = 0.0;
        for (int k = 0; k < K; k++)
        {
            const int i = trans_A ? (k * M + row) : (row * K + k);
            const int j = trans_B ? (col * K + k) : (k * N + col);
            sum += A[i] * B[j];
        }

        C[row * N + col]  = sum;
    }
}

std::vector<torch::Tensor> mylinear_cuda_forward(
    torch::Tensor input,
    torch::Tensor weights)
{
    const int M = input.size(0);
    const int K = input.size(1);
    const int N = weights.size(0);

    auto output = torch::zeros({M, N}, torch::TensorOptions().device(torch::kCUDA));

    const dim3 block(32, 32);
    const dim3 grid((M - 1) / 32 + 1, (N - 1) / 32 + 1);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "mylinear_cuda_forward", ([&] {
        matmul_kernel<scalar_t><<<grid, block>>>(
            input.data<scalar_t>(),
            weights.data<scalar_t>(),
            output.data<scalar_t>(),
            M,
            K,
            N,
            false,
            true);
        }));
    
    return {output};
}

std::vector<torch::Tensor> mylinear_cuda_backward(
    torch::Tensor grad_output,
    torch::Tensor input,
    torch::Tensor weights)
{
    const int M = grad_output.size(0);
    const int N = grad_output.size(1);
    const int K = weights.size(1);

    auto grad_input = torch::zeros({M, K}, torch::TensorOptions().device(torch::kCUDA));
    auto grad_weights = torch::zeros({N, K}, torch::TensorOptions().device(torch::kCUDA));

    const dim3 block(32, 32);
    const dim3 grid1((M - 1) / 32 + 1, (K - 1) / 32 + 1);
    const dim3 grid2((N - 1) / 32 + 1, (K - 1) / 32 + 1);


    AT_DISPATCH_FLOATING_TYPES(input.type(), "mylinear_cuda_backward_input", ([&] {
        matmul_kernel<scalar_t><<<grid1, block>>>(
            grad_output.data<scalar_t>(),
            weights.data<scalar_t>(),
            grad_input.data<scalar_t>(),
            M,
            N,
            K,
            false,
            false);
        }));

    AT_DISPATCH_FLOATING_TYPES(input.type(), "mylinear_cuda_backward_input", ([&] {
        matmul_kernel<scalar_t><<<grid2, block>>>(
            grad_output.data<scalar_t>(),
            input.data<scalar_t>(),
            grad_weights.data<scalar_t>(),
            N,
            M,
            K,
            true,
            false);
        }));
    
    return {grad_input, grad_weights};
}